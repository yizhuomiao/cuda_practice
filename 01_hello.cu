// @file main.cu


#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void hello_world(){
    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    printf("[ %d ] Hello world from GPU\n", idx);
}
int main() {
	printf("Hello world from CPU\n");

    hello_world<<< 1, 10 >>>();
    hipDeviceSynchronize();
    return 0;
}
